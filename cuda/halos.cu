#include "hip/hip_runtime.h"
#include "../comms.h"
#include "../mesh.h"
#include "halos.k"
#include "shared.h"

// Enforce reflective boundary conditions on the problem state
void handle_boundary(
    const int nx, const int ny, Mesh* mesh, double* arr, 
    const int invert, const int prepare)
{
  START_PROFILING(&comms_profile);

  int* neighbours = mesh->neighbours;

#ifdef MPI
  int nmessages = 0;

  if(prepare) {
    // prepare east and west
    if(neighbours[EAST] != EDGE) {
      int nblocks = ceil(ny*PAD/(double)NTHREADS);
      prepare_east<<<nblocks, NTHREADS>>>(
          nx, ny, mesh->east_buffer_out, arr);

      sync_data(ny*PAD, &mesh->east_buffer_out, &mesh->h_east_buffer_out, RECV);
      non_block_send(mesh->h_east_buffer_out, ny*PAD, neighbours[EAST], 2, nmessages++);
      non_block_recv(mesh->h_east_buffer_in, ny*PAD, neighbours[EAST], 3, nmessages++);
    }

    if(neighbours[WEST] != EDGE) {
      int nblocks = ceil(ny*PAD/(double)NTHREADS);
      prepare_west<<<nblocks, NTHREADS>>>(
          nx, ny, mesh->west_buffer_out, arr);

      sync_data(ny*PAD, &mesh->west_buffer_out, &mesh->h_west_buffer_out, RECV);
      non_block_send(mesh->h_west_buffer_out, ny*PAD, neighbours[WEST], 3, nmessages++);
      non_block_recv(mesh->h_west_buffer_in, ny*PAD, neighbours[WEST], 2, nmessages++);
    }

    // prepare north and south
    if(neighbours[NORTH] != EDGE) {
      int nblocks = ceil(nx*PAD/(double)NTHREADS);
      prepare_north<<<nblocks, NTHREADS>>>(
          nx, ny, mesh->north_buffer_out, arr);

      sync_data(nx*PAD, &mesh->north_buffer_out, &mesh->h_north_buffer_out, RECV);
      non_block_send(mesh->h_north_buffer_out, nx*PAD, neighbours[NORTH], 1, nmessages++);
      non_block_recv(mesh->h_north_buffer_in, nx*PAD, neighbours[NORTH], 0, nmessages++);
    }

    if(neighbours[SOUTH] != EDGE) {
      int nblocks = ceil(nx*PAD/(double)NTHREADS);
      prepare_south<<<nblocks, NTHREADS>>>(
          nx, ny, mesh->south_buffer_out, arr);

      sync_data(nx*PAD, &mesh->south_buffer_out, &mesh->h_south_buffer_out, RECV);
      non_block_send(mesh->h_south_buffer_out, nx*PAD, neighbours[SOUTH], 0, nmessages++);
      non_block_recv(mesh->h_south_buffer_in, nx*PAD, neighbours[SOUTH], 1, nmessages++);
    }

    wait_on_messages(nmessages);

    // Unprepare east and west
    if(neighbours[WEST] != EDGE) {
      sync_data(ny*PAD, &mesh->h_west_buffer_in, &mesh->west_buffer_in, SEND);

      int nblocks = ceil(ny*PAD/(double)NTHREADS);
      retrieve_west<<<nblocks, NTHREADS>>>(
          nx, ny, mesh->west_buffer_in, arr);
    }

    if(neighbours[EAST] != EDGE) {
      sync_data(ny*PAD, &mesh->h_east_buffer_in, &mesh->east_buffer_in, SEND);

      int nblocks = ceil(ny*PAD/(double)NTHREADS);
      retrieve_east<<<nblocks, NTHREADS>>>(
          nx, ny, mesh->east_buffer_in, arr);
    }

    // Unprepare north and south
    if(neighbours[NORTH] != EDGE) {
      sync_data(nx*PAD, &mesh->h_north_buffer_in, &mesh->north_buffer_in, SEND);

      int nblocks = ceil(nx*PAD/(double)NTHREADS);
      retrieve_north<<<nblocks, NTHREADS>>>(
          nx, ny, mesh->north_buffer_in, arr);
    }

    if(neighbours[SOUTH] != EDGE) {
      sync_data(nx*PAD, &mesh->h_south_buffer_in, &mesh->south_buffer_in, SEND);

      int nblocks = ceil(nx*PAD/(double)NTHREADS);
      retrieve_south<<<nblocks, NTHREADS>>>(
          nx, ny, mesh->south_buffer_in, arr);
    }
  }
#endif

  // Perform the boundary reflections, potentially with the data updated from neighbours
  double x_inversion_coeff = (invert == INVERT_X) ? -1.0 : 1.0;
  double y_inversion_coeff = (invert == INVERT_Y) ? -1.0 : 1.0;

  // Reflect at the north
  if(neighbours[NORTH] == EDGE) {
    int nblocks = ceil(nx*PAD/(double)NTHREADS);
    north_boundary<<<nblocks, NTHREADS>>>(
        nx, ny, y_inversion_coeff, arr);
  }
  // reflect at the south
  if(neighbours[SOUTH] == EDGE) {
    int nblocks = ceil(nx*PAD/(double)NTHREADS);
    south_boundary<<<nblocks, NTHREADS>>>(
        nx, ny, y_inversion_coeff, arr);
  }
  // reflect at the east
  if(neighbours[EAST] == EDGE) {
    int nblocks = ceil(ny*PAD/(double)NTHREADS);
    east_boundary<<<nblocks, NTHREADS>>>(
        nx, ny, x_inversion_coeff, arr);
  }
  // reflect at the west
  if(neighbours[WEST] == EDGE) {
    int nblocks = ceil(ny*PAD/(double)NTHREADS);
    west_boundary<<<nblocks, NTHREADS>>>(
        nx, ny, x_inversion_coeff, arr);
  }
  STOP_PROFILING(&comms_profile, __func__);

  gpu_check(hipDeviceSynchronize());
}

