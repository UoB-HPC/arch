#include "hip/hip_runtime.h"
#include <omp.h>
#include <stdio.h>
#include <unistd.h>
#include "../cuda/shared.h"

#define NDBLOCKS 3
#define NVARIABLES 2
#define GB (1024LLU*1024LLU*1024LLU)

__global__ void init(const int n, int* x, int* y);
__global__ void compute(const int n, int* x, int* y);
__global__ void validate(const int n, int* x, int* y);

int main()
{
  // Calculate the blocks locally
  const size_t pages = sysconf(_SC_PHYS_PAGES);
  const size_t page_size = sysconf(_SC_PAGE_SIZE);
  size_t free_dram_memory = pages*page_size;

  // Fudging a value in for testing
  const size_t ablock_bytes = free_dram_memory*0.8;
  const size_t ablock_space_len = ablock_bytes/sizeof(int);
  int* ablock_space = (int*)malloc(ablock_bytes);
  printf("DRAM Memory Capacity Available. Free = %llu\n", free_dram_memory);
  int* ablock[NVARIABLES];
  const size_t ablock_len = ablock_space_len/NVARIABLES;
  for(int vv = 0; vv < NVARIABLES; ++vv) {
    ablock[vv] = &ablock_space[vv*ablock_len];
  }
  printf("Application Data Block Length %llu\n", ablock_len);

  // Work out how large data staging blocks are available
  size_t free_gpu_mem, total_gpu_mem;
  hipMemGetInfo(&free_gpu_mem, &total_gpu_mem);

  // Fudging a value in for testing
  printf("GPU Memory Capacity Available. Free = %llu\n", free_gpu_mem, total_gpu_mem);
  const size_t dblock_bytes = (free_gpu_mem*0.8)/(NVARIABLES*NDBLOCKS);
  const size_t max_dblock_len = dblock_bytes/sizeof(int);
  printf("Max Data Block Length %llu\n", max_dblock_len);

  // Allocate a validation bit for the validation routine
  int* validation_bit;
  hipMalloc((void**)&validation_bit, 1);
  gpu_check(hipDeviceSynchronize());

  // Initialise all of the data staging blocks
  int* dblocks[NVARIABLES][NDBLOCKS];
  for(int vv = 0; vv < NVARIABLES; ++vv) {
    for(int dd = 0; dd < NDBLOCKS; ++dd) {
      hipMalloc((void**)&dblocks[vv][dd], dblock_bytes);
      gpu_check(hipDeviceSynchronize());
    }
  }

  // Create streams for asynchronous copies
  hipStream_t in_stream;
  hipStream_t out_stream;
  gpu_check(hipStreamCreate(&in_stream));
  gpu_check(hipStreamCreate(&out_stream));

  /* BEGIN STAGIN ROUTINE */
  const size_t ndblocks_reqd = ceil(ablock_len/(double)max_dblock_len);
  printf("Requiring %llu Data Blocks\n", ndblocks_reqd);

  for(int ii = 0; ii < ndblocks_reqd; ++ii) {
    const size_t in_id =  (ii+2)%NDBLOCKS;
    const size_t on_id =  (ii+1)%NDBLOCKS;
    const size_t out_id = (ii+0)%NDBLOCKS;
    const size_t dblock_len = ((ii+1)*max_dblock_len > ablock_len) 
      ? ablock_len-ii*max_dblock_len : max_dblock_len;

    printf("Stagin: in %llu on %llu out %llu dblock_len %llu\n", 
        in_id, on_id, out_id, dblock_len);

    // Copy on the first block into the 'on' data staging block
    if(ii == 0) {
      for(int vv = 0; vv < NVARIABLES; ++vv) {
        hipMemcpyAsync(
            dblocks[vv][on_id], 
            &ablock[vv][0],
            max_dblock_len*sizeof(int), hipMemcpyHostToDevice, in_stream);
      }

      // Have to sync here to prepare initial data
      gpu_check(hipStreamSynchronize(in_stream));
      gpu_check(hipDeviceSynchronize());
    }

    // If not last iteration, asynchronously stage new blocks
    if(ii < ndblocks_reqd-1) {
      const size_t next_dblock_len = ((ii+2)*max_dblock_len > ablock_len) 
        ? ablock_len-ii*max_dblock_len : max_dblock_len;
      for(int vv = 0; vv < NVARIABLES; ++vv) {
        hipMemcpyAsync(
            dblocks[vv][in_id],
            &ablock[vv][(ii+1)*max_dblock_len],
            next_dblock_len*sizeof(int), hipMemcpyHostToDevice, in_stream);
      }
    }
    gpu_check(hipStreamSynchronize(in_stream));
    gpu_check(hipDeviceSynchronize());

    // Perform the operation
    const size_t nblocks = ceil(dblock_len/(double)NTHREADS);
    init<<<nblocks, NTHREADS>>>(dblock_len, dblocks[0][on_id], dblocks[1][on_id]);
    gpu_check(hipDeviceSynchronize());

    // After first iteration, begin copying blocks back
    if(ii > 0) {
      for(int vv = 0; vv < NVARIABLES; ++vv) {
        hipMemcpyAsync(
            &ablock[vv][(ii-1)*max_dblock_len],
            dblocks[vv][out_id],
            max_dblock_len*sizeof(int), hipMemcpyDeviceToHost, out_stream);
      }
    }

    gpu_check(hipDeviceSynchronize());
    gpu_check(hipStreamSynchronize(out_stream));

    // Copy back the last computed block
    if(ii == ndblocks_reqd-1) {
      for(int vv = 0; vv < NVARIABLES; ++vv) {
        hipMemcpyAsync(
            &ablock[vv][ii*max_dblock_len],
            dblocks[vv][on_id],
            dblock_len*sizeof(int), hipMemcpyDeviceToHost, out_stream);
      }
    }

    gpu_check(hipStreamSynchronize(out_stream));
    gpu_check(hipDeviceSynchronize());
  }

  for(int ii = 0; ii < ablock_len; ++ii) {
    if(ablock[0][ii] != 1) {
      printf("0 %d unsuccessful initialisation %d\n", ii, ablock[0][ii]);
    }
#if 0
    else {
      printf("%d successful initialisation %d\n", ii, ablock[0][ii]);
    }
#endif // if 0
    if(ablock[1][ii] != 2) {
      printf("1 %d unsuccessful initialisation %d\n", ii, ablock[1][ii]);
    }
#if 0
    else {
      printf("%d successful initialisation %d\n", ii, ablock[1][ii]);
    }
#endif // if 0
  }

#if 0
  /* BEGIN STAGIN ROUTINE */
  const size_t ndblocks_reqd = ceil(ablock_len/max_dblock_len);
  for(int ii = 0; ii < ndblocks_reqd; ++ii) {

    const int in_id =  (ii+0)%NDBLOCKS;
    const int on_id =  (ii+1)%NDBLOCKS;
    const int out_id = (ii+2)%NDBLOCKS;

    // If not last iteration, stage new blocks
    if(ii < ndblocks_reqd-1) {
      for(int vv = 0; vv < NVARIABLES; ++vv) {
        hipMemcpyAsync(
            &dblocks[in_id*NVARIABLES+vv], &(ablock[ii*max_dblock_len]), max_dblock_len,
            hipMemcpyHostToDevice, in_stream);
      }
    }

    compute<<<nblocks, NTHREADS>>>(
        max_dblock_len, dblocks[on_id*NVARIABLES+0], dblocks[on_id*NVARIABLES+1]);
    gpu_check(hipDeviceSynchronize());

    // After first iteration, begin copying blocks back
    if(ii > 0) {
      for(int vv = 0; vv < NVARIABLES; ++vv) {
        hipMemcpyAsync(
            &(ablock[ii*max_dblock_len]), &dblocks[in_id*NVARIABLES+vv], max_dblock_len,
            hipMemcpyDeviceToHost, out_stream);
      }
    }
  }

  /* BEGIN STAGIN ROUTINE */
  const size_t ndblocks_reqd = ceil(ablock_len/max_dblock_len);
  for(int ii = 0; ii < ndblocks_reqd; ++ii) {

    const int in_id =  (ii+0)%NDBLOCKS;
    const int on_id =  (ii+1)%NDBLOCKS;
    const int out_id = (ii+2)%NDBLOCKS;

    // If not last iteration, stage new blocks
    if(ii < ndblocks_reqd-1) {
      for(int vv = 0; vv < NVARIABLES; ++vv) {
        hipMemcpyAsync(
            &dblocks[in_id*NVARIABLES+vv], &(ablock[ii*max_dblock_len]), max_dblock_len,
            hipMemcpyHostToDevice, in_stream);
      }
    }

    validate<<<nblocks, NTHREADS>>>(max_dblock_len, dblocks[0], validation_bit);
    gpu_check(hipDeviceSynchronize());

    // After first iteration, begin copying blocks back
    if(ii > 0) {
      for(int vv = 0; vv < NVARIABLES; ++vv) {
        hipMemcpyAsync(
            &(ablock[ii*max_dblock_len]), &dblocks[in_id*NVARIABLES+vv], max_dblock_len,
            hipMemcpyDeviceToHost, out_stream);
      }
    }
  }

  int l_validation_bit;
  int* h_validation_bit = &l_validation_bit;
  hipMemcpy(&h_validation_bit, &validation_bit, 1, hipMemcpyDeviceToHost);
  if(l_validation_bit == 99) {
    printf("Failed validation\n");
  }
  else {
    printf("Successful validation\n");
  }
#endif // if 0

  gpu_check(hipStreamDestroy(in_stream));
  gpu_check(hipStreamDestroy(out_stream));
}

__global__ void init(const int n, int* x, int* y)
{
  const int gid = blockDim.x*blockIdx.x+threadIdx.x;
  if(gid < n) {
    x[gid] = 1;
    y[gid] = 2;
  }
}

__global__ void compute(const int n, int* x, int* y)
{
  const int gid = blockDim.x*blockIdx.x+threadIdx.x;
  if(gid < n) {
    x[gid] = x[gid]*y[gid];
  }
}

__global__ void validate(const int n, int* x, int* y)
{
  const int gid = blockDim.x*blockIdx.x+threadIdx.x;
  if(gid < n) {
    if(x[gid] != 2) {
      y[0] = 99;
    }
  }
}

